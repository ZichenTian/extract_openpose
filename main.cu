#include <iostream>
#include <array>
#include "utils.h"

void nmsGpuFloatWrapper(
    float* dst, int* kernel, float* src, const float threshold, 
    const std::array<int, 4>& dstSize, const std::array<int, 4>& srcSize, const Point<float>& offset);

template <typename T>
void initSrcData(int n, int c, int h, int w, T* data) {
    const size_t size = n * c * h * w;      // 输入数据
    for(size_t i = 0; i < size; i++) {
        data[i] = i; 
    }
}

template <typename T>
void showDstData(int n, int c, int h, int w, const T* data) {
    const size_t size = n * c * h * w;      // 输出数据
    for(size_t i = 0; i < size; i++) {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;
}

template <typename T>
void nmsWrapper(void) {

    /*********************** 参数设置 *******************************/
    const int batch = 1;            // 这些参数是我从srcBase.h里面扒的
    const int srcChannels = 57;     // 具体意思我不太清楚
    const int srcHeight = 368;
    const int srcWidth = 656;
    std::array<int, 4> srcSize = {batch, srcChannels, srcHeight, srcWidth};

    const int dstChannels = 18;
    const int dstHeight = 128;
    const int dstWidth = 3;
    std::array<int, 4> dstSize = {batch, dstChannels, dstHeight, dstWidth};

    Point<T> offset;              // 这个参数不知道该咋设
    offset.x = 0;
    offset.y = 0;

    const T threshold = 0.5;        // 这个参数是我乱设的，不清楚行不行

    const size_t srcLength = batch * srcChannels * srcHeight * srcWidth;
    const size_t kernelLength = batch * srcChannels * srcHeight * srcWidth; // same as srcSize
    const size_t dstLength = batch * dstChannels * dstHeight * dstWidth;

    /**************************** 存储空间分配 ***************************/

    T* srcDataCpu = new T[srcLength];
    initSrcData<T>(batch, srcChannels, srcHeight, srcWidth, srcDataCpu);

    T* srcDataCuda;
    int* kernelDataCuda;
    T* dstDataCuda;
    hipMalloc((void**)&srcDataCuda, srcLength * sizeof(T));
    hipMalloc((void**)&kernelDataCuda, kernelLength * sizeof(int));
    hipMalloc((void**)&dstDataCuda, dstLength * sizeof(T));

    T* dstDataCpu = new T[dstLength];
    int* kernelDataCpu = nullptr;

    /*************************** NMS运算 ********************************/

    hipMemcpy(srcDataCuda, srcDataCpu, srcLength * sizeof(T), hipMemcpyHostToDevice);     // cpu copy to gpu

    nmsGpuFloatWrapper(dstDataCpu, kernelDataCuda, srcDataCpu, threshold, dstSize, srcSize, offset);       // compute
    hipDeviceSynchronize();                                                                        // wait for compute finished

    hipMemcpy(dstDataCpu, dstDataCuda, dstLength * sizeof(T), hipMemcpyDeviceToHost);     // copy back to cpu

    showDstData(batch, dstChannels, dstHeight, dstWidth, dstDataCpu);

    /******************************* 释放空间 **********************************/

    hipFree(srcDataCuda);
    hipFree(kernelDataCuda);
    hipFree(dstDataCuda);

    delete[] srcDataCpu;
    delete[] dstDataCpu;
}

int main(int argc, char* argv[]) {
    nmsWrapper<float>();
    return 0;
}
